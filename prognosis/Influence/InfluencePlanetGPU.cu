//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// find force from planet
//==============================================================================//
#include <stdio.h>
#include "InfluenceForce.h"

#ifdef GPUCOMPILE

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// name
namespace Force
{
	//==============================================================================//
	// ��������� ������
	// �������� �������� �� GPU
	//==============================================================================//
	void InfluenceForce::cuDE403LoadFileToGPU()
	{
		printf("Load DE403 to GPU\n");
		int sizeMem = (2668442 + 10000)*sizeof( double );

		printf( "GPU MemSize = %d B\n", sizeMem );
		hipError_t error;
		error = hipMalloc( (void**)&d_FileMemDE403, sizeMem );
		if (error != hipSuccess) {
			printf("Failed hipMalloc (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		error = hipMemcpy( d_FileMemDE403, FileMemDE403, sizeMem, hipMemcpyHostToDevice );
		if (error != hipSuccess) {
			printf("Failed hipMalloc (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
	}
}
//==============================================================================//

#endif

