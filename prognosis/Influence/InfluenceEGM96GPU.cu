//==============================================================================//
// Andrianov N.G.
// opbit predict 
// module find Influence Force
// find force from Earth harmonics
// copy array to gpu
//==============================================================================//
#include <math.h>
#include <fstream>
#include <stdio.h>
#include "InfluenceForce.h"

#ifdef GPUCOMPILE

#include "cutilNP.h"
#include "hip/hip_runtime.h"
#include ""

namespace Force
{
	//==============================================================================//
	// Load To GPU
	//==============================================================================//
	void InfluenceForce::cuLoadegm96ToGPU()
	{
		int memoryEgm = 6561*sizeof( double );
		cutilSafeCall( hipMalloc( (void**)&d_egm96, memoryEgm ) );
		cutilSafeCall( hipMemcpy( d_egm96, h_egm96, memoryEgm, hipMemcpyHostToDevice )) ;
	};
	//==============================================================================//
};

#endif